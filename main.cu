#include "hip/hip_runtime.h"
#include "materials.hpp"

#include "timer.hpp"

#include <random>
#include <thread>
#include <iostream>

namespace compiler {
static void please_dont_optimize_away([[maybe_unused]] void* p) { asm volatile("" : : "g"(p) : "memory"); }
}

double random_real() { 
  static std::default_random_engine generator;
  static std::uniform_real_distribution<double> distribution(-1.0, 1.0);
  return distribution(generator);
}

void check_answers(const std::vector < J2MaterialData > & A, 
                   const std::vector < J2MaterialData > & B) {

  int n = A.size();

  double norm[4] = {};
  double error[4] = {};
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < BLOCK_SIZE; j++) {
      for (int row = 0; row < 3; row++) {
        for (int col = 0; col < 3; col++) {
          {
            double value = A[i].F_old[row][col][j];
            double diff = B[i].F_old[row][col][j] - value;
            norm[0] += value * value;
            error[0] += diff * diff;
            if (diff * diff > 1.0e-10) {
              printf("%d, %d, %d, %d, %d, %f\n", 0, i, j, row, col, diff);
            }
          }

          {
            double value = A[i].be_bar[row][col][j];
            double diff = B[i].be_bar[row][col][j] - value;
            norm[1] += value * value;
            error[1] += diff * diff;
            if (diff * diff > 1.0e-10) {
              printf("%d, %d, %d, %d, %d, %f\n", 1, i, j, row, col, diff);
            }
          }

          {
            double value = A[i].tau[row][col][j];
            double diff = B[i].tau[row][col][j] - value;
            norm[2] += value * value;
            error[2] += diff * diff;
            if (diff * diff > 1.0e-10) {
              printf("%d, %d, %d, %d, %d, %f\n", 2, i, j, row, col, diff);
            }
          }
        }
      }

      double value = A[i].alpha[j];
      double diff = B[i].alpha[j] - value;
      norm[3] += value * value;
      error[3] += diff * diff;
      if (diff * diff > 1.0e-10) {
        printf("%d, %d, %d, %f\n", 2, i, j, diff);
      }
    }
  }
  if (sqrt(error[0] / norm[0]) > 1.0e-14) {
    std::cout << "relative frobenius error (F):      " << sqrt(error[0] / norm[0]) << std::endl;
  }
  if (sqrt(error[1] / norm[1]) > 1.0e-14) {
    std::cout << "relative frobenius error (be_bar): " << sqrt(error[1] / norm[1]) << std::endl;
  }
  if (sqrt(error[2] / norm[2]) > 1.0e-14) {
    std::cout << "relative frobenius error (tau):    " << sqrt(error[2] / norm[2]) << std::endl;
  }
  if (sqrt(error[3] / norm[3]) > 1.0e-14) {
    std::cout << "relative frobenius error (alpha):  " << sqrt(error[3] / norm[3]) << std::endl;
  }
}

__global__ void J2_kernel(J2MaterialData * data, int n) {

  int i = threadIdx.y + blockIdx.x * blockDim.y;
  if (i < n) {
    J2MaterialDataScalar tdata;

    tdata.K = data[i].K[threadIdx.x];
    tdata.mu = data[i].mu[threadIdx.x];
    tdata.sigma_y = data[i].sigma_y[threadIdx.x];
    tdata.alpha = data[i].alpha[threadIdx.x];
    for (int r = 0; r < 3; r++) {
      for (int c = 0; c < 3; c++) {
        tdata.F_old(r,c) = data[i].F_old[r][c][threadIdx.x];
        tdata.F_new(r,c) = data[i].F_new[r][c][threadIdx.x];
        tdata.be_bar_old(r,c) = data[i].be_bar[r][c][threadIdx.x];
      }
    }

    J2_plasticity_model_cuda(tdata); 

    data[i].alpha[threadIdx.x] = tdata.alpha;
    for (int r = 0; r < 3; r++) {
      for (int c = 0; c < 3; c++) {
        data[i].F_old[r][c][threadIdx.x] = tdata.F_new(r,c);
        data[i].tau[r][c][threadIdx.x] = tdata.tau(r,c);
        data[i].be_bar[r][c][threadIdx.x] = tdata.be_bar_old(r,c);
      }
    }

  }
  
}

void J2_plasticity_test(int n, int num_runs) {

  timer stopwatch;

  std::vector < J2MaterialData > data(n);

  std::cout << "J2 plasticity model comparison test" << std::endl;
  std::cout << "  generating input data ... " << std::endl;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < BLOCK_SIZE; j++) {
      data[i].K[j]  = 100.0 + 20.0 * random_real();
      data[i].mu[j] = 100.0 + 20.0 * random_real();
      data[i].sigma_y[j] = 0.1;
      data[i].alpha[j] = 1.0 + random_real();
      for (int row = 0; row < 3; row++) {
        for (int col = 0; col < 3; col++) {
          data[i].F_old[row][col][j] = (row == col) + 0.1 * random_real();
          data[i].F_new[row][col][j] = (row == col) + 0.1 * random_real();
          
          double value = 0.1 * random_real();
          data[i].be_bar[row][col][j] = value;
          data[i].be_bar[col][row][j] = value;
        }
      }
    } 
  }

  std::vector < J2MaterialData > data_copy = data;

  std::cout << "                    naive J2 implementation: ";
  stopwatch.start();
  for (int k = 0; k < num_runs; k++) {
    for (int i = 0; i < n; i++) {
      J2_plasticity_model_scalar(data[i]);
    }
    compiler::please_dont_optimize_away(&data);
  }
  stopwatch.stop();
  std::cout << stopwatch.elapsed() / num_runs << "s per run" << std::endl;

  auto answers = data;

  data = data_copy;

  std::cout << "               vectorized J2 implementation: "; 
  stopwatch.start();
  for (int k = 0; k < num_runs; k++) {
    for (int i = 0; i < n; i++) {
      J2_plasticity_model_simd(data[i]);
    }
    compiler::please_dont_optimize_away(&data);
  }
  stopwatch.stop();
  std::cout << stopwatch.elapsed() / num_runs << "s per run" << std::endl;

  check_answers(answers, data);

  data = data_copy;

  int num_threads = std::thread::hardware_concurrency();
  std::cout << "  " << num_threads << " threads + vectorized J2 implementation: "; 
  stopwatch.start();
  for (int k = 0; k < num_runs; k++) {
    std::vector< std::thread > threads;
    for (int i = 0; i < num_threads; i++) {
      threads.push_back(std::thread([=, &data](){
        for (int k = i; k < n; k+=num_threads) {
          J2_plasticity_model_simd(data[k]);
        }
      }));
    }
    for (int i = 0; i < num_threads; i++) {
      threads[i].join();
    }
    compiler::please_dont_optimize_away(&data);
  }
  stopwatch.stop();
  std::cout << stopwatch.elapsed() / num_runs << "s per run" << std::endl;

  check_answers(answers, data);

  std::cout << "                     cuda J2 implementation: "; 
  J2MaterialData * data_d;
  hipMalloc(&data_d, sizeof(J2MaterialData) * n);
  hipMemcpy(data_d, &data_copy[0], sizeof(J2MaterialData) * n, hipMemcpyHostToDevice);

  stopwatch.start();
  for (int k = 0; k < num_runs; k++) {
    dim3 block {BLOCK_SIZE, 4, 1};
    dim3 grid {uint32_t(n) / 4, 1, 1};
    J2_kernel<<<grid, block>>>(data_d, n);
    compiler::please_dont_optimize_away(&data_d);
  }
  hipDeviceSynchronize();
  stopwatch.stop();
  std::cout << stopwatch.elapsed() / num_runs << "s per run" << std::endl;

  hipMemcpy(&data[0], data_d, sizeof(J2MaterialData) * n, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipFree(data_d);

  check_answers(answers, data);

}

int main(int argc, char *argv[]) {

  int n = 1000000;
  int num_runs = 5;

  if (argc >= 2) {
    n = atoi(argv[1]);
  }

  //int n = 1 << 8;

  //neohookean_test(n, num_runs);
  J2_plasticity_test(n, num_runs);

}
