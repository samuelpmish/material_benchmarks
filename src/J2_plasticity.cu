#include "hip/hip_runtime.h"
#include "materials.hpp"

#include "enoki.hpp" // defines vdouble, load(), store()
#include "matrix.hpp"

__device__ void J2_plasticity_model_cuda(J2MaterialDataScalar & data) {

  using mat3 = matrix<double, 3, 3>;

  static constexpr auto I = Identity<3>();

  // compute elastic predictor
  mat3 fbar_new = dot(data.F_new, inv(data.F_old));
  fbar_new /= cbrt(det(fbar_new));
  mat3 be_bar_new = dot(fbar_new, dot(data.be_bar_old, transpose(fbar_new)));
  double Ie_bar_new = tr(be_bar_new) / 3.0;
  mat3 s_new = data.mu * dev(be_bar_new);

  // check for plastic loading
  double y_new = norm(s_new) - sqrt(2.0 / 3.0) * (data.K * data.alpha + data.sigma_y);

  // return mapping algorithm
  if (y_new > 0) {
    double mu_bar = data.mu / Ie_bar_new;

    double delta_gamma = (3 * y_new) / (2 * data.K + 6 * mu_bar);

    mat3 n = s_new / norm(s_new);
    s_new -= 2 * mu_bar * delta_gamma * n;

    data.alpha += sqrt(2.0 / 3.0) * delta_gamma;
  }
  
  double J_new = det(data.F_new);
  double p_new = (0.5 * data.K) * (J_new - (1.0 / J_new));

  data.tau = (J_new * p_new) * I + s_new;
  data.be_bar_old = (s_new / data.mu) + Ie_bar_new * I;

}
